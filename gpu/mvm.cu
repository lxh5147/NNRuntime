#include "mvm.cuh"
#include <assert.h>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 4
#endif

template<typename T>
static void mvm_gpu(const T* A, const T* x, T* y, const size_t row, const size_t col){
    assert(A);
    assert(x);
    assert(y);
    assert(row>0);
    assert(col>0);
    T* A_d;
    hipMalloc((void**)&A_d,sizeof(T)*row*col);
    hipMemcpy(A_d, A, sizeof(T)*row*col, hipMemcpyHostToDevice);
    T* x_d;
    hipMalloc((void**)&x_d,sizeof(T)*row);
    hipMemcpy(x_d, x, sizeof(T)*row, hipMemcpyHostToDevice);
    T* y_d;
    hipMalloc((void**)&y_d,sizeof(T)*row);
    int numberOfBlocks=(row+BLOCK_SIZE-1)/BLOCK_SIZE;
    mvm_kernal<<<numberOfBlocks,BLOCK_SIZE>>> (A_d,x_d,y_d,row,col,BLOCK_SIZE);
    hipMemcpy(y,y_d,sizeof(T)*row, hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(x_d);
    hipFree(y_d);
}

#define IMPL_MVM_GPU_FUNC(T) extern "C" void mvm_gpu(const ##T* A, const ##T* x, ##T* y, const size_t row, const size_t col) {mvm_gpu<##T>(A,x,y,row,col)}

IMPL_MVM_GPU_FUNC(float);
IMPL_MVM_GPU_FUNC(double);

#endif
